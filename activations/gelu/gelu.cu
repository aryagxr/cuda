#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <math.h>
#include <iostream>


/*  Kernel 1: FP32 */
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define sqrt2overPI 0.7978845608
#define k 0.044715

__global__ void kernel1_gelu_fp32(float* in, float* out, int n){
    int tidx = threadIdx.x + (blockDim.x * blockIdx.x);

    if(tidx < n){
        float x = in[tidx];
        out[tidx] = 0.5f * x * (1.0f + tanh(sqrt2overPI * (x + k * (x * x * x))));
    }
}


/*  Kernel 2: Vectorized FP32 * 4 */
__global__ void kernel2_gelu_4fp32_vectorized(float* in, float* out, int n){
    int tidx = (threadIdx.x + (blockDim.x * blockIdx.x)) * 4;

    if(tidx < n){
        float4 x = FLOAT4(in[tidx]);
        float4 y;
        y.x = 0.5f * x.x * (1.0f + tanh(sqrt2overPI * (x.x + k * (x.x * x.x * x.x))));
        y.y = 0.5f * x.y * (1.0f + tanh(sqrt2overPI * (x.y + k * (x.y * x.y * x.y))));
        y.z = 0.5f * x.z * (1.0f + tanh(sqrt2overPI * (x.z + k * (x.z * x.z * x.z))));
        y.w = 0.5f * x.w * (1.0f + tanh(sqrt2overPI * (x.w + k * (x.w * x.w * x.w))));

        FLOAT4(out[tidx]) = y;
    }
}


/* Kernel 3: Half precision*/
__global__ void kernel3_gelu_fp16(half* in, half* out, int n){
    int tidx = threadIdx.x + (blockDim.x * blockIdx.x);

    if(tidx < n){
        half x = in[tidx];
        float xf = __half2float(x);
        float tanhx = sqrt2overPI * (xf + k * (xf * xf * xf));
        float tanhv = tanh(tanhx);
        half tanh_half = __float2half(tanhv);
        out[tidx] = __hmul(__float2half(0.5f), __hmul(x, __hadd(__float2half(1.0f), tanh_half)));
    }
}



int main(){
    const int N = 1024;
    size_t fp32_size = N * sizeof(float);

    float *X, *P, *P2;
    float *dx, *dp, *dp2;
    
    X = (float*)malloc(fp32_size);
    P = (float*)malloc(fp32_size);
    P2 = (float*)malloc(fp32_size);
    

    hipMalloc((void**)&dx, fp32_size);
    hipMalloc((void**)&dp, fp32_size);
    hipMalloc((void**)&dp2, fp32_size);
    
    
    
    for (int i = 0; i < N; ++i) {
        X[i] = (rand() / float(RAND_MAX)) * 2.0f - 1.0f;
    }

    std::cout << "Input matrix:" << std::endl;
    for (int i = 0; i < 10; ++i) {
        std::cout << X[i] << " ";
    }
    std::cout << std::endl;

    hipMemcpy(dx, X, fp32_size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock-1) / threadsPerBlock;
    int blocksPerGrid_vec = (N / 4 + threadsPerBlock - 1) / threadsPerBlock;
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms = 0.f;
    hipEventRecord(start);
    

    kernel1_gelu_fp32<<<blocksPerGrid, threadsPerBlock>>>(dx, dp, N);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Scalar FP32 Kernel1 execution time: " << ms << " ms\n";

    hipMemcpy(P, dp, fp32_size, hipMemcpyDeviceToHost);


    // Run vectorized FP32 * 4 kernel
    hipEventRecord(start);
    kernel2_gelu_4fp32_vectorized<<<blocksPerGrid_vec, threadsPerBlock>>>(dx, dp2, N);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Vectorized FP32*4 Kernel2 execution time: " << ms << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(P2, dp2, fp32_size, hipMemcpyDeviceToHost);
    


    for (int i = 0; i < 10; ++i) {
        std::cout << P[i] << " ";
    }
    std::cout << std::endl;

    for (int i = 0; i < 10; ++i) {
        std::cout << P2[i] << " ";
    }


    hipFree(dx); hipFree(dp); hipFree(dp2);
    free(X); free(P); free(P2);

    return 0;
    
}

