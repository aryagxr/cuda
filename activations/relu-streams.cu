#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>


#define N 1024
#define NSTREAMS 4


/* 1 block per stream */
__global__ void relu_streams(float* in, float* out, int n, int offset){
    int tidx = threadIdx.x + blockDim.x * blockIdx.x + offset;

    if(tidx < offset + n){
        out[tidx] = fmaxf(0.0, in[tidx]);
    }
    __syncthreads();
}



int main(){

    size_t vec_size = N * sizeof(float);
    int streamSize = N / NSTREAMS;
    int streamBytes = streamSize * sizeof(float);

    int threads = 256;
    int blocks = (streamSize + threads - 1) / threads;

    // pinned memory
    float *hx, *hp;
    hipHostMalloc(&hx, vec_size, hipHostMallocDefault);
    hipHostMalloc(&hp, vec_size, hipHostMallocDefault);
    float *dx, *dp;
    hipMalloc((void**)&dx, vec_size);
    hipMalloc((void**)&dp, vec_size);

    for (int i = 0; i < N; ++i) {
        hx[i] = (rand() / float(RAND_MAX)) * 2.0f - 1.0f;
    }

    std::cout << "First 10 elements of Input: ";
    for (int i = 0; i < 10; i++) {
        std::cout << hx[i] << " ";
    }
    std::cout << std::endl;

    hipStream_t stream[NSTREAMS];
    for(int i = 0; i < NSTREAMS; i++){
        hipStreamCreate(&stream[i]);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    /* Async Version 1: Copy, Kernel, Copy per stream*/
    for(int i = 0; i < NSTREAMS; i++){

        int offset = i * streamSize;

        // H2D
        hipMemcpyAsync(&dx[offset], &hx[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);

        // kernel
        relu_streams<<<blocks, threads, 0, stream[i]>>>(dx, dp, streamSize, offset);


        // D2H
        hipMemcpyAsync(&hp[offset], &dp[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);

    }

    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamSynchronize(stream[i]);
        hipStreamDestroy(stream[i]);
    }


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Time taken for kernel execution: " << elapsedTime << " ms" << std::endl;


    std::cout << "First 10 elements of Output: ";
    for (int i = 0; i < 10; i++) {
        std::cout << hp[i] << " ";
    }
    std::cout << std::endl;

    hipFree(dx); hipFree(dp);
    hipHostFree(hx); hipHostFree(hp);

    return 0;
}