#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <iostream>

#define N 5

// deriv = 1 for x>=0, 0 for x<0
// dout -> upstream gradients (wrt loss)
// din -> output of backpass of relu
__global__ void relu_backpass(float* out, float* din, float* dout, int n) {
    int tidx = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tidx < n) {
        float x = out[tidx];
        din[tidx] = (x >= 0) ? dout[tidx] : 0.0f;
    }
}

int main() {
    float h_in[N] = {-2.0, 0.0, 1.5, -3.0, 4.0};   // Forward input
    float h_dout[N] = {0.1, -0.2, 0.5, 0.3, -0.7}; // Incoming gradient
    float h_out[N] = {0.0, 0.0, 1.5, 0.0, 4.0};    // Sample forward pass output

    float *d_in, *d_out, *d_dout, *d_din;
    float h_din[N];


    hipMalloc((void**)&d_in, N * sizeof(float));
    hipMalloc((void**)&d_out, N * sizeof(float));
    hipMalloc((void**)&d_dout, N * sizeof(float));
    hipMalloc((void**)&d_din, N * sizeof(float));

    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, N * sizeof(float), hipMemcpyHostToDevice); // Use sample forward pass output
    hipMemcpy(d_dout, h_dout, N * sizeof(float), hipMemcpyHostToDevice);

    std::cout << "Forward Pass Output (ReLU):\n";
    for (int i = 0; i < N; i++) std::cout << h_out[i] << " ";
    std::cout << "\n";


    relu_backpass<<<1, N>>>(d_out, d_din, d_dout, N);
    hipDeviceSynchronize();
    hipMemcpy(h_din, d_din, N * sizeof(float), hipMemcpyDeviceToHost);


    std::cout << "Backward Pass Input (h_dout):\n";
    for (int i = 0; i < N; i++) std::cout << h_dout[i] << " ";
    std::cout << "\n";

    std::cout << "Backward Pass Output (Gradient h_din):\n";
    for (int i = 0; i < N; i++) std::cout << h_din[i] << " ";
    std::cout << "\n";


    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_dout);
    hipFree(d_din);

    return 0;
}