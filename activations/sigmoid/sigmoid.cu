#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <iostream>

#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

__global__ void kernel1_sigmoid_fp32(float* in, float* out, int n){
    int tidx = threadIdx.x + (blockDim.x * blockIdx.x);
    if(tidx < n){
        float x = in[tidx];
        out[tidx] = 1 / (1 + expf(-x));
    }
}

__global__ void kernel2_sigmoid_4fp32(float* in, float* out, int n){
    int tidx = (threadIdx.x + (blockDim.x * blockIdx.x)) * 4;
    if(tidx < n){
        float4 x = FLOAT4(in[tidx]);
        float4 y;
        y.x = 1 / (1 + expf(-x.x));
        y.y = 1 / (1 + expf(-x.y));
        y.z = 1 / (1 + expf(-x.z));
        y.w = 1 / (1 + expf(-x.w));

        FLOAT4(out[tidx]) = y;
    }
}

__global__ void kernel3_sigmoid_fp16(half* in, half* out, int n){
    int tidx = threadIdx.x + (blockDim.x * blockIdx.x);
    if(tidx < n) {
        half x = in[tidx];
        float x_float = __half2float(x);
        float result_float = 1.0f / (1.0f + expf(-x_float));
        out[tidx] = __float2half(result_float);
    }
}




int main(){
    const int N = 1024;
    size_t fp32_size = N * sizeof(float);
    size_t fp16_size = N * sizeof(half);

    float *X, *P, *P2;
    float *dx, *dp, *dp2;
    half *X_half;
    float *P3;
    half *dx_half, *dp_half;
    
    X = (float*)malloc(fp32_size);
    P = (float*)malloc(fp32_size);
    P2 = (float*)malloc(fp32_size);
    P3 = (float*)malloc(fp32_size);
    X_half = (half*)malloc(fp16_size);

    hipMalloc((void**)&dx, fp32_size);
    hipMalloc((void**)&dp, fp32_size);
    hipMalloc((void**)&dp2, fp32_size);
    hipMalloc((void**)&dx_half, fp16_size);
    hipMalloc((void**)&dp_half, fp16_size);
    
    for (int i = 0; i < N; ++i) {
        X[i] = (rand() / float(RAND_MAX)) * 2.0f - 1.0f;
        X_half[i] = __float2half(X[i]);
    }


    hipMemcpy(dx, X, fp32_size, hipMemcpyHostToDevice);
    hipMemcpy(dx_half, X_half, fp16_size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock-1) / threadsPerBlock;
    int blocksPerGrid_vec = (N / 4 + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms = 0.f;
    hipEventRecord(start);
    

    kernel1_sigmoid_fp32<<<blocksPerGrid, threadsPerBlock>>>(dx, dp, N);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Scalar FP32 Kernel1 execution time: " << ms << " ms\n";

    hipMemcpy(P, dp, fp32_size, hipMemcpyDeviceToHost);


    // Run vectorized FP32 * 4 kernel
    hipEventRecord(start);
    kernel2_sigmoid_4fp32<<<blocksPerGrid_vec, threadsPerBlock>>>(dx, dp2, N);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Vectorized FP32*4 Kernel2 execution time: " << ms << " ms\n";


    hipEventRecord(start);
    kernel3_sigmoid_fp16<<<blocksPerGrid, threadsPerBlock>>>(dx_half, dp_half, N);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "FP16 Kernel3 execution time: " << ms << " ms\n";
    

    hipEventDestroy(start);
    hipEventDestroy(stop);


    hipMemcpy(P2, dp2, fp32_size, hipMemcpyDeviceToHost);
    half* P_half = (half*)malloc(fp16_size);
    hipMemcpy(P_half, dp_half, fp16_size, hipMemcpyDeviceToHost);

    
    for (int i = 0; i < N; ++i) {
        P3[i] = __half2float(P_half[i]);
    }


    for (int i = 0; i < 10; ++i) {
        std::cout << P[i] << " ";
    }
    std::cout << std::endl;

    for (int i = 0; i < 10; ++i) {
        std::cout << P2[i] << " ";
    }

    std::cout << "\n\nFirst 10 results from FP16 kernel:\n";
    for (int i = 0; i < 10; ++i) {
        std::cout << P3[i] << " ";
    }
    std::cout << std::endl;


    hipFree(dx); hipFree(dp); hipFree(dp2); hipFree(dx_half); hipFree(dp_half);
    free(X); free(P); free(P2); free(P3); free(X_half); free(P_half);


    return 0;
    
}



