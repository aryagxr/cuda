#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <iostream>

__global__ void kernel1_swish_fp32(float* in, float* out, int n){
    int tidx = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tidx < n){
        float x = in[tidx];
        out[tidx] = x / (1.0f + expf(-x));
    }
}


int main() {
    const int N = 1024 * 1024;
    const size_t bytes = N * sizeof(float);
    
    float *h_input = (float*)malloc(bytes);
    float *h_output = (float*)malloc(bytes);
    
    srand(42);
    for (int i = 0; i < N; ++i) {
        h_input[i] = (rand() / float(RAND_MAX)) * 2.0f - 1.0f;
    }
    
    float *d_input, *d_output;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);
    
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    kernel1_swish_fp32<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost);
    
    std::cout << "Swish kernel execution time: " << milliseconds << " ms" << std::endl;
    std::cout << "Throughput: " << (N * sizeof(float)) / (milliseconds * 1.0e6) << " GB/s" << std::endl;
    
    std::cout << "\nSample results:" << std::endl;
    for (int i = 0; i < 10; ++i) {
        std::cout << "Input: " << h_input[i] << " → Swish: " << h_output[i] << std::endl;
    }
    
    std::cout << "\nVerification (CPU vs GPU):" << std::endl;
    bool verification_passed = true;
    for (int i = 0; i < 5; ++i) {
        float cpu_result = h_input[i] / (1.0f + expf(-h_input[i]));
        float error = fabs(cpu_result - h_output[i]);
        std::cout << "CPU: " << cpu_result << " GPU: " << h_output[i];
        std::cout << " (error: " << error << ")" << std::endl;
        if (error > 1e-5) {
            verification_passed = false;
        }
    }
    std::cout << "\nVerification " << (verification_passed ? "PASSED" : "FAILED") << std::endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    
    return 0;
}