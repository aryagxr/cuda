#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void fa1_forward(float *Q, float *K, float*V, float *O, float *l, float *m, int N, int Br, int Bc, int Tr, int Tc, int d, float scale){

    int tidx = threadIdx.x;

    // on chip smem for K,V
    __shared__ float K_tile[Bc][d];
    __shared__ float V_tile[Bc][d];
    __shared__ float Q_tile[Br][d]; 
    __shared__ float O_tile[Br][d];
    __shared__ float l_tile[Br];
    __shared__ float m_tile[Br];


    int row_offset = blockIdx.x * Br;
    int col_offset = blockIdx.y * Bc;

    //  5:  for 1 ≤ 𝑗 ≤ 𝑇𝑐 d
    for(int j = 1; j < Tc; j++){

        int k_offset = j * Bc * d;
        int v_offset = j * Bc * d;

        // 6: Load K𝑗, V𝑗 from HBM to on-chip SRA
        for (int i = tidx; i < Bc * d; i += blockDim.x) {
            int row = i / d;
            int col = i % d;
            K_tile[row][col] = K[k_offset + i]; // Load from HBM (global memory)
            V_tile[row][col] = V[v_offset + i];
        }
        __syncthreads();


        // 7:  for 1 ≤ 𝑖 ≤ 𝑇𝑟 d
        for(int i = 1; i < Tr; i++){

            int Q_offset = i * Br * d;
            int O_offset = i * Br * d;
            int l_offset = i * Br;
            int m_offset = i * Br;

            // 8:  Load Q𝑖, O𝑖, ℓ𝑖, 𝑚𝑖 from HBM to on-chip SRAM
            for(int t = tidx; t < Br * d; t += blockDim.x){
                int row = t/d;
                int col = t%d;
                Q_tile[row][col] = Q[Q_offset + t];
                O_tile[row][col] = O[O_offset + t];
            }

            for (int t = tidx; t < Br; t += blockDim.x) {
                l_tile[t] = l[l_offset + t];
                m_tile[t] = m[m_offset + t];
            }
            __syncthreads();
        


        }



        

    }


}