#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void SpMV_CSR(int rows, float *data, int *col_idx, int *row_ptr, float *x, float *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows) {
        float dot = 0.0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row+1];
        
        for (int i = row_start; i < row_end; i++) {
            dot += data[i] * x[col_idx[i]];
        }
        
        y[row] += dot;
    }
}


int main() {
    const int rows = 4;
    const int nnz = 9; // number of non-zero elements

    float h_data[nnz] = {10, 20, 30, 40, 50, 60, 70, 80, 90};
    int h_col_idx[nnz] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int h_row_ptr[rows + 1] = {0, 3, 6, 7, 9};

    float h_x[3] = {1, 2, 3};
    float h_y[rows] = {0, 0, 0, 0};

    float *d_data, *d_x, *d_y;
    int *d_col_idx, *d_row_ptr;
    hipMalloc(&d_data, nnz * sizeof(float));
    hipMalloc(&d_col_idx, nnz * sizeof(int));
    hipMalloc(&d_row_ptr, (rows + 1) * sizeof(int));
    hipMalloc(&d_x, 3 * sizeof(float));
    hipMalloc(&d_y, rows * sizeof(float));

    hipMemcpy(d_data, h_data, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, h_col_idx, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, h_row_ptr, (rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, rows * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (rows + blockSize - 1) / blockSize;
    SpMV_CSR<<<numBlocks, blockSize>>>(rows, d_data, d_col_idx, d_row_ptr, d_x, d_y);

    hipMemcpy(h_y, d_y, rows * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result vector y:" << std::endl;
    for (int i = 0; i < rows; i++) {
        std::cout << h_y[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_data);
    hipFree(d_col_idx);
    hipFree(d_row_ptr);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}