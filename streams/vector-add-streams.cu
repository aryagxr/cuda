#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>


#define N 1024
#define NSTREAMS 4

__global__ void vect_add_streams(float* a, float* b, float* c, int offset, int n){

    //int tidx = threadIdx.x + (blockDim.x * blockIdx.x);
    int tidx = threadIdx.x + blockIdx.x * blockDim.x + offset;
    if(tidx < offset + n){
        c[tidx] = a[tidx] + b[tidx];
    }

}


int main(){
    
    int size = N * sizeof(float);
    const int threadsPerBlock = 256;
    const int streamSize = N / NSTREAMS;
    int streamBytes = streamSize * sizeof(float);
    int blocksPerGrid = (streamSize + threadsPerBlock - 1) / threadsPerBlock;

    // Using pinned host memory
    float *ha, *hb, *hc;
    hipHostMalloc(&ha, size, hipHostMallocDefault);
    hipHostMalloc(&hb, size, hipHostMallocDefault);
    hipHostMalloc(&hc, size, hipHostMallocDefault);

    float *da, *db, *dc;
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);

    for (int i = 0; i < N; i++) {
        ha[i] = static_cast<float>(i);
        hb[i] = static_cast<float>(i * 2);
    }

    std::cout << "First 10 elements of A: ";
    for (int i = 0; i < 10; i++) {
        std::cout << ha[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "First 10 elements of B: ";
    for (int i = 0; i < 10; i++) {
        std::cout << hb[i] << " ";
    }
    std::cout << std::endl;



    hipStream_t stream[NSTREAMS];
    for(int i = 0; i < NSTREAMS; i++){
        hipStreamCreate(&stream[i]);
    }


    /* Async Version 1: Copy, Kernel, Copy per stream*/
    for(int i = 0; i < NSTREAMS; i++){
        int offset = streamSize * i;

        // copy: H2D
        hipMemcpyAsync(&da[offset], &ha[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&db[offset], &hb[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);

        // kernel
        vect_add_streams<<<blocksPerGrid, threadsPerBlock, 0, stream[i]>>>(da, db, dc, offset, streamSize);

        // copy: D2H
        hipMemcpyAsync(&hc[offset], &dc[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);

    }

    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamSynchronize(stream[i]);
        hipStreamDestroy(stream[i]);
    }

    std::cout << "First 10 elements of C: ";
    for (int i = 0; i < 10; i++) {
        std::cout << hc[i] << " ";
    }
    std::cout << std::endl;

    bool success = true;
    for (int i = 0; i < N; i++) {
        if (hc[i] != ha[i] + hb[i]) {
            success = false;
            std::cerr << "Mismatch at index " << i << std::endl;
            break;
        }
    }
    std::cout << (success ? "Success!" : "Failed!") << std::endl;

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipHostFree(ha);
    hipHostFree(hb);
    hipHostFree(hc);

    return 0;

}
