#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>



using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;



/*  Kernel 1: Naive implementation of Matrix Transpose
    Using shared memory, with bank conflicts
*/
__global__ void kernel1_naive_smem_mattrans(float *A, float *A_T, int m, int n){

    __shared__ float smem[32][32];

    int row = threadIdx.y + (blockDim.y * blockIdx.y);
    int col = threadIdx.x + (blockDim.x * blockIdx.x);

    if(row < m && col < n){

        //load 32*32 tile into shared memory
        // each block has its own smem
        // blockdim = tile dim
        smem[threadIdx.x][threadIdx.y] = A[row * n + col];
        __syncthreads();

        // bank conflict happens here
        A_T[col * m + row] = smem[threadIdx.x][threadIdx.y];

    }
}


/*  Kernel 2: Matrix Transpose using shared memory & padding
    (without swizzling)
*/
__global__ void kernel2_smem_padding_mattrans(float *A, float *A_T, int m, int n){

    __shared__ float smem[32][33]; //extra padding col

    int row = threadIdx.y + (blockDim.y * blockIdx.y);
    int col = threadIdx.x + (blockDim.x * blockIdx.x);

    if(row < m && col < n){

        smem[threadIdx.x][threadIdx.y] = A[row * n + col];
        __syncthreads();
        A_T[col * m + row] = smem[threadIdx.x][threadIdx.y];

    }
}



/*  Kernel 3: Matrix Transpose with Swizzling
    XOR is used to swizzle smem indices
*/
__global__ void kernel3_swizzled_mattrans(float *A, float *A_T, int m, int n){

    __shared__ float smem[32][32];
    int row = threadIdx.y + (blockDim.y * blockIdx.y);
    int col = threadIdx.x + (blockDim.x * blockIdx.x);
    if(row < m && col < n){

        
        smem[threadIdx.x][threadIdx.x ^ threadIdx.y] = A[row * n + col];
        __syncthreads();

        A_T[col * m + row] = smem[threadIdx.x][threadIdx.x ^ threadIdx.y];

    }

}




int main(){
    const int M = 1024;
    const int N = 1024;
    size_t mat_size = M*N*sizeof(float);
    float *hA, *hA_t;
    float *dA, *dA_t;

    hA = (float*)malloc(mat_size);
    hA_t = (float*)malloc(mat_size);

    for(int i=0; i<M*N; i++){
        hA[i]=i+1;
    }

    hipMalloc((void**)&dA, mat_size);
    hipMalloc((void**)&dA_t, mat_size);

    hipMemcpy(dA, hA, mat_size, hipMemcpyHostToDevice);


    dim3 threadsPerBlock(32,32); 
    dim3 blocksPerGrid(N/2, M/2); 

    kernel3_swizzled_mattrans<<<blocksPerGrid, threadsPerBlock>>>(dA, dA_t, M, N);
    hipDeviceSynchronize();

    hipMemcpy(hA_t, dA_t, mat_size, hipMemcpyDeviceToHost);

    /*
    for (int j = 0; j < 10; j++) {
        for (int i = 0; i < 10; i++) {
            printf("%f ", hA_t[j * M + i]);
        }
        printf("\n");
    }
    printf("Successful");
    */

    hipFree(dA); hipFree(dA_t);
    free(hA); free(hA_t);

}